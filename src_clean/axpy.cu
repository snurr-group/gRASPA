#include "hip/hip_runtime.h"
#include "axpy.h"
#include "mc_single_particle.h"
#include "mc_swap_moves.h"
#include "mc_box.h"

#include "write_data.h"

#include "print_statistics.cuh"

//#include "lambda.h"
#include <numeric>
#include <cmath>
#include <algorithm>
#include <filesystem>
#include <optional>

#include <fstream>

inline void Copy_AtomData_from_Device(Atoms* System, Atoms* d_a, Components& SystemComponents, Boxsize& HostBox, Simulations& Sims)
{
  hipMemcpy(System, d_a, SystemComponents.Total_Components * sizeof(Atoms), hipMemcpyDeviceToHost);
  for(size_t ijk=0; ijk < SystemComponents.Total_Components; ijk++)
  {
    if(SystemComponents.HostSystem[ijk].Allocate_size != System[ijk].Allocate_size)
    {
      // if the host allocate_size is different from the device, allocate more space on the host
      SystemComponents.HostSystem[ijk].pos       = (double3*) malloc(System[ijk].Allocate_size*sizeof(double3));
      SystemComponents.HostSystem[ijk].scale     = (double*)  malloc(System[ijk].Allocate_size*sizeof(double));
      SystemComponents.HostSystem[ijk].charge    = (double*)  malloc(System[ijk].Allocate_size*sizeof(double));
      SystemComponents.HostSystem[ijk].scaleCoul = (double*)  malloc(System[ijk].Allocate_size*sizeof(double));
      SystemComponents.HostSystem[ijk].Type      = (size_t*)  malloc(System[ijk].Allocate_size*sizeof(size_t));
      SystemComponents.HostSystem[ijk].MolID     = (size_t*)  malloc(System[ijk].Allocate_size*sizeof(size_t));
      SystemComponents.HostSystem[ijk].Allocate_size = System[ijk].Allocate_size;
    }
  
    hipMemcpy(SystemComponents.HostSystem[ijk].pos, System[ijk].pos, sizeof(double3)*System[ijk].Allocate_size, hipMemcpyDeviceToHost);
    hipMemcpy(SystemComponents.HostSystem[ijk].scale, System[ijk].scale, sizeof(double)*System[ijk].Allocate_size, hipMemcpyDeviceToHost);
    hipMemcpy(SystemComponents.HostSystem[ijk].charge, System[ijk].charge, sizeof(double)*System[ijk].Allocate_size, hipMemcpyDeviceToHost);
    hipMemcpy(SystemComponents.HostSystem[ijk].scaleCoul, System[ijk].scaleCoul, sizeof(double)*System[ijk].Allocate_size, hipMemcpyDeviceToHost);
    hipMemcpy(SystemComponents.HostSystem[ijk].Type, System[ijk].Type, sizeof(size_t)*System[ijk].Allocate_size, hipMemcpyDeviceToHost);
    hipMemcpy(SystemComponents.HostSystem[ijk].MolID, System[ijk].MolID, sizeof(size_t)*System[ijk].Allocate_size, hipMemcpyDeviceToHost);
    SystemComponents.HostSystem[ijk].size = System[ijk].size;
  }
  HostBox.Cell = (double*) malloc(9 * sizeof(double));
  HostBox.InverseCell = (double*) malloc(9 * sizeof(double));
  hipMemcpy(HostBox.Cell,        Sims.Box.Cell,        sizeof(double)*9, hipMemcpyDeviceToHost);
  hipMemcpy(HostBox.InverseCell, Sims.Box.InverseCell, sizeof(double)*9, hipMemcpyDeviceToHost);
  HostBox.Cubic = Sims.Box.Cubic;
}

inline void GenerateRestartMovies(Components& SystemComponents, Simulations& Sims, PseudoAtomDefinitions& PseudoAtom, size_t systemIdx, int SimulationMode)
{
  //Generate Restart file during the simulation, regardless of the phase
  Atoms device_System[SystemComponents.NComponents.x];
  Boxsize HostBox;
  Copy_AtomData_from_Device(device_System, Sims.d_a, SystemComponents, HostBox, Sims);
  create_Restart_file(0, SystemComponents.HostSystem, SystemComponents, SystemComponents.FF, HostBox, PseudoAtom.Name, systemIdx);
  Write_All_Adsorbate_data(0, SystemComponents.HostSystem, SystemComponents, SystemComponents.FF, HostBox, PseudoAtom.Name, systemIdx);
  //Only generate LAMMPS data movie for production phase
  if(SimulationMode == PRODUCTION)  create_movie_file(SystemComponents.HostSystem, SystemComponents, HostBox, PseudoAtom.Name, systemIdx);
}

///////////////////////////////////////////////////////////
// Wrapper for Performing a move for the selected system //
///////////////////////////////////////////////////////////
inline void RunMoves(int Cycle, Components& SystemComponents, Simulations& Sims, ForceField& FF, RandomNumber& Random, WidomStruct& Widom, double& Rosenbluth, int SimulationMode)
{
  SystemComponents.CURRENTCYCLE = Cycle;
  //Randomly Select an Adsorbate Molecule and determine its Component: MoleculeID --> Component
  //Zhao's note: The number of atoms can be vulnerable, adding throw error here//
  if(SystemComponents.TotalNumberOfMolecules < SystemComponents.NumberOfFrameworks)
    throw std::runtime_error("There is negative number of adsorbates. Break program!");

  size_t comp = 0; // When selecting components, skip the component 0 (because it is the framework)
  size_t SelectedMolecule = 0; 
  size_t SelectedMolInComponent = 0;

  size_t NumberOfImmobileFrameworkMolecules = 0; size_t ImmobileFrameworkSpecies = 0;
  for(size_t i = 0; i < SystemComponents.NComponents.y; i++)
    if(SystemComponents.Moves[i].TotalProb < 1e-10)
    {
      ImmobileFrameworkSpecies++;
      NumberOfImmobileFrameworkMolecules += SystemComponents.NumberOfMolecule_for_Component[i];
    }
  while(SystemComponents.Moves[comp].TotalProb < 1e-10)
  {
    comp = (size_t) (Get_Uniform_Random() * SystemComponents.NComponents.x);
  }
  SelectedMolInComponent = (size_t) (Get_Uniform_Random() * SystemComponents.NumberOfMolecule_for_Component[comp]);

  MoveEnergy DeltaE;
  double RANDOMNUMBER = Get_Uniform_Random();
  //printf("Step %zu, selected Comp %zu, Mol %zu, RANDOM: %.5f", Cycle, comp, SelectedMolInComponent, RANDOMNUMBER);
  if(RANDOMNUMBER < SystemComponents.Moves[comp].TranslationProb)
  {
    //////////////////////////////
    // PERFORM TRANSLATION MOVE //
    //////////////////////////////
    //printf(" Translation\n");
    if(SystemComponents.NumberOfMolecule_for_Component[comp] > 0)
    {
      DeltaE = SingleBodyMove(SystemComponents, Sims, Widom, FF, Random, SelectedMolInComponent, comp, TRANSLATION);
    }
    else
    {
      SystemComponents.Tmmc[comp].Update(1.0, SystemComponents.NumberOfMolecule_for_Component[comp], TRANSLATION);
    }
  }
  else if(RANDOMNUMBER < SystemComponents.Moves[comp].RotationProb) //Rotation
  {
    ///////////////////////////
    // PERFORM ROTATION MOVE //
    ///////////////////////////
    //printf(" Rotation\n");
    if(SystemComponents.NumberOfMolecule_for_Component[comp] > 0)
    {
      DeltaE = SingleBodyMove(SystemComponents, Sims, Widom, FF, Random, SelectedMolInComponent, comp, ROTATION);
    }
    else
    {
      SystemComponents.Tmmc[comp].Update(1.0, SystemComponents.NumberOfMolecule_for_Component[comp], ROTATION);
    }
  }
  else if(RANDOMNUMBER < SystemComponents.Moves[comp].SpecialRotationProb) //Special Rotation for Framework Components
  {
    ///////////////////////////////////
    // PERFORM SPECIAL ROTATION MOVE //
    ///////////////////////////////////
    //printf(" Special Rotation\n");
    if(SystemComponents.NumberOfMolecule_for_Component[comp] > 0)
      DeltaE = SingleBodyMove(SystemComponents, Sims, Widom, FF, Random, SelectedMolInComponent, comp, SPECIAL_ROTATION);
  }
  else if(RANDOMNUMBER < SystemComponents.Moves[comp].WidomProb)
  {
    //////////////////////////////////
    // PERFORM WIDOM INSERTION MOVE //
    //////////////////////////////////
    //printf(" Widom Insertion\n");
    double2 newScale = SystemComponents.Lambda[comp].SET_SCALE(1.0); //Set scale for full molecule (lambda = 1.0)//
    /*
    size_t SelectedTrial=0; bool SuccessConstruction = false; MoveEnergy energy; double StoredR = 0.0;
    double Rosenbluth=Widom_Move_FirstBead_PARTIAL(SystemComponents, Sims, FF, Random, Widom, SelectedMolInComponent, comp, CBMC_INSERTION, StoredR, &SelectedTrial, &SuccessConstruction, &energy, newScale);
    if(SystemComponents.Moleculesize[comp] > 1 && Rosenbluth > 1e-150)
    {
      size_t SelectedFirstBeadTrial = SelectedTrial;
      Rosenbluth*=Widom_Move_Chain_PARTIAL(SystemComponents, Sims, FF, Random, Widom, SelectedMolInComponent, comp, CBMC_INSERTION, &SelectedTrial, &SuccessConstruction, &energy, SelectedFirstBeadTrial, newScale); 
      //Zhao's note: need to add widom insertion data back//
    }
    */
    double Rosenbluth = WidomMove(SystemComponents, Sims, FF, Random, Widom, SelectedMolInComponent, comp, newScale);
    SystemComponents.Moves[comp].RecordRosen(Rosenbluth, WIDOM);
  }
  else if(RANDOMNUMBER < SystemComponents.Moves[comp].ReinsertionProb)
  {
    //////////////////////////////
    // PERFORM REINSERTION MOVE //
    //////////////////////////////
    //printf(" Reinsertion\n");
    if(SystemComponents.NumberOfMolecule_for_Component[comp] > 0)
    {
      DeltaE = Reinsertion(SystemComponents, Sims, FF, Random, Widom, SelectedMolInComponent, comp);
    }
    else
    {
      SystemComponents.Tmmc[comp].Update(1.0, SystemComponents.NumberOfMolecule_for_Component[comp], REINSERTION);
    }
  }
  else if(RANDOMNUMBER < SystemComponents.Moves[comp].IdentitySwapProb)
  {
    //printf(" Identity Swap\n");
    DeltaE = IdentitySwapMove(SystemComponents, Sims, Widom, FF, Random);
  }
  else if(RANDOMNUMBER < SystemComponents.Moves[comp].CBCFProb && SystemComponents.hasfractionalMolecule[comp])
  {
    ///////////////////////
    // PERFORM CBCF MOVE //
    ///////////////////////
    //printf(" CBCF\n");
    SelectedMolInComponent = SystemComponents.Lambda[comp].FractionalMoleculeID;
    DeltaE = CBCFMove(SystemComponents, Sims, FF, Random, Widom, SelectedMolInComponent, comp);
  }
  else if(RANDOMNUMBER < SystemComponents.Moves[comp].SwapProb)
  {
    ////////////////////////////
    // PERFORM GCMC INSERTION //
    ////////////////////////////
    if(Get_Uniform_Random() < 0.5)
    {
      //printf(" Swap Insertion\n");
      if(!SystemComponents.SingleSwap)
      {
        DeltaE = Insertion(SystemComponents, Sims, FF, Random, Widom, SelectedMolInComponent, comp);
      }
      else
      {
        DeltaE = SingleBodyMove(SystemComponents, Sims, Widom, FF, Random, SelectedMolInComponent, comp, SINGLE_INSERTION);
        //DeltaE = SingleSwapMove(SystemComponents, Sims, Widom, FF, Random, SelectedMolInComponent, comp, SINGLE_INSERTION);
      }
    }
    else
    {
      ///////////////////////////
      // PERFORM GCMC DELETION //
      ///////////////////////////
      //printf(" Swap Deletion\n");
      //Zhao's note: Do not do a deletion if the chosen molecule is a fractional molecule, fractional molecules should go to CBCFSwap moves//
      if(!((SystemComponents.hasfractionalMolecule[comp]) && SelectedMolInComponent == SystemComponents.Lambda[comp].FractionalMoleculeID))
      {
        if(SystemComponents.NumberOfMolecule_for_Component[comp] > 0)
        {
          if(!SystemComponents.SingleSwap)
          {
            DeltaE = Deletion(SystemComponents, Sims, FF, Random, Widom, SelectedMolInComponent, comp);
          }
          else
          {
            //DeltaE = SingleSwapMove(SystemComponents, Sims, Widom, FF, Random, SelectedMolInComponent, comp, SINGLE_DELETION);
            DeltaE = SingleBodyMove(SystemComponents, Sims, Widom, FF, Random, SelectedMolInComponent, comp, SINGLE_DELETION);
          }
        }
        else
        {
          SystemComponents.Tmmc[comp].Update(0.0, SystemComponents.NumberOfMolecule_for_Component[comp], DELETION);
        }
      }
    }
  }
  
  if(Cycle == 4)
  {
    printf("Cycle [%d], Printing DeltaE\n", Cycle);
    DeltaE.print();
  }
  
  SystemComponents.deltaE += DeltaE;
}

double CreateMolecule_InOneBox(Components& SystemComponents, Simulations& Sims, ForceField FF, RandomNumber& Random, WidomStruct Widom, bool AlreadyHasFractionalMolecule)
{
  double running_energy = 0.0;
  // Create Molecules in the Box Before the Simulation //
  for(size_t comp = SystemComponents.NComponents.y; comp < SystemComponents.NComponents.x; comp++)
  {
    size_t CreateFailCount = 0; size_t Created = 0; size_t SelectedMol = 0;
    CreateFailCount = 0;
    printf("Component %zu, Need to create %zu full molecule\n", comp, SystemComponents.NumberOfCreateMolecules[comp]);
    //Create Fractional Molecule first//
    if(SystemComponents.hasfractionalMolecule[comp])
    {
      //Zhao's note: If we need to create fractional molecule, then we initialize WangLandau Histogram//
      size_t FractionalMolToCreate = 1;
      if(AlreadyHasFractionalMolecule) FractionalMolToCreate = 0;
      if(FractionalMolToCreate > 0) Initialize_WangLandauIteration(SystemComponents.Lambda[comp]);
      while(FractionalMolToCreate > 0)
      {
        printf("Creating Fractional Molecule for Component %zu; There are %zu Molecules of that component in the System\n", comp, SystemComponents.NumberOfMolecule_for_Component[comp]);
        SelectedMol = Created; if(Created > 0) SelectedMol = Created - 1; 
        //Zhao's note: this is a little confusing, but when number of molecule for that species = 0 or 1, the chosen molecule is zero. This is creating from zero loading, need to change in the future, when we read from restart file//
        size_t OldVal = SystemComponents.NumberOfMolecule_for_Component[comp];

        size_t NewBin = 5;
        if(SystemComponents.Tmmc[comp].DoTMMC) NewBin = 0;
        double newLambda = static_cast<double>(NewBin) * SystemComponents.Lambda[comp].delta;
        double2 newScale = SystemComponents.Lambda[comp].SET_SCALE(newLambda);
        MoveEnergy DeltaE;
        DeltaE = CreateMolecule(SystemComponents, Sims, FF, Random, Widom, SelectedMol, comp, newScale);
        running_energy += DeltaE.total();
        SystemComponents.CreateMoldeltaE += DeltaE;
        if(SystemComponents.NumberOfMolecule_for_Component[comp] == OldVal)
        {
          CreateFailCount ++;
        }
        else
        {
          FractionalMolToCreate --; Created ++; SystemComponents.Lambda[comp].FractionalMoleculeID = SelectedMol;
          SystemComponents.Lambda[comp].currentBin = NewBin;
        }
        if(CreateFailCount > 1000000000) throw std::runtime_error("Bad Insertions When Creating Fractional Molecules!");
      }
    }
    while(SystemComponents.NumberOfCreateMolecules[comp] > 0)
    {
      printf("Creating %zu Molecule for Component %zu; There are %zu Molecules of that component in the System\n", Created, comp, SystemComponents.NumberOfMolecule_for_Component[comp]);
      SelectedMol = Created; if(Created > 0) SelectedMol = Created - 1; //Zhao's note: this is a little confusing, but when number of molecule for that species = 0 or 1, the chosen molecule is zero. This is creating from zero loading, need to change in the future, when we read from restart file//
      size_t OldVal    = SystemComponents.NumberOfMolecule_for_Component[comp];
      double2 newScale = SystemComponents.Lambda[comp].SET_SCALE(1.0); //Set scale for full molecule (lambda = 1.0)//
      MoveEnergy DeltaE;
      DeltaE = CreateMolecule(SystemComponents, Sims, FF, Random, Widom, SelectedMol, comp, newScale);
      //printf("Creating %zu molecule\n", SelectedMol);
      //DeltaE.print();
      running_energy += DeltaE.total();
      SystemComponents.CreateMoldeltaE += DeltaE;
      printf("Delta E in creating molecules:\n"); DeltaE.print();
      if(SystemComponents.NumberOfMolecule_for_Component[comp] == OldVal)
      {CreateFailCount ++;} else {SystemComponents.NumberOfCreateMolecules[comp] --; Created ++;}
      if(CreateFailCount > 10000) throw std::runtime_error("Bad Insertions When Creating Molecules!");
    }
  }
  return running_energy;
}

void Run_Simulation_MultipleBoxes(int Cycles, std::vector<Components>& SystemComponents, Simulations*& Sims, ForceField FF, RandomNumber& Random, std::vector<WidomStruct>& Widom, std::vector<SystemEnergies>& Energy, Gibbs& GibbsStatistics, int SimulationMode, bool SetMaxStep, size_t MaxStepPerCycle)
{
  size_t NumberOfSimulations = SystemComponents.size();
  size_t WLSampled = 0; size_t WLAdjusted = 0;

  std::vector<int> BlockAverageSize(NumberOfSimulations, 1);

  std::string Mode;
  switch(SimulationMode)
  {
    case INITIALIZATION:{Mode = "INITIALIZATION"; break;}
    case EQUILIBRATION: {Mode = "EQUILIBRATION"; break;}
    case PRODUCTION:    {Mode = "PRODUCTION"; break;}
  } 
  /*
  if(SimulationMode == INITIALIZATION && Cycles > 0)
  {
    for(size_t sim = 0; sim < NumberOfSimulations; sim++)
    { 
      SystemComponents[sim].deltaVDWReal = 0.0;
      SystemComponents[sim].deltaEwald   = 0.0;
    }
  }
  */
  if(SimulationMode == PRODUCTION)
  {
    for(size_t sim = 0; sim < NumberOfSimulations; sim++)
    {
      BlockAverageSize[sim] = Cycles / SystemComponents[sim].Nblock;
      if(Cycles % SystemComponents[sim].Nblock != 0)
        printf("Warning! Number of Cycles cannot be divided by Number of blocks. Residue values go to the last block\n");
      SystemComponents[sim].BookKeepEnergy.resize(SystemComponents[sim].Nblock);
      SystemComponents[sim].BookKeepEnergy_SQ.resize(SystemComponents[sim].Nblock);
    }
  }

  std::vector<double> running_Rosenbluth(NumberOfSimulations, 0.0);

  /////////////////////////////////////////////
  // FINALIZE (PRODUCTION) CBCF BIASING TERM //
  /////////////////////////////////////////////
  if(SimulationMode == PRODUCTION)
  {
    for(size_t sim = 0; sim < NumberOfSimulations; sim++)
      for(size_t icomp = 0; icomp < SystemComponents[sim].Total_Components; icomp++)
        if(SystemComponents[sim].hasfractionalMolecule[icomp] && !SystemComponents[sim].Tmmc[icomp].DoTMMC)
          Finalize_WangLandauIteration(SystemComponents[sim].Lambda[icomp]);
  }

  ///////////////////////////////////////////////////////////////////////
  // FORCE INITIALIZING CBCF BIASING TERM BEFORE INITIALIZATION CYCLES //
  ///////////////////////////////////////////////////////////////////////
  if(SimulationMode == INITIALIZATION && Cycles > 0)
  {
    for(size_t sim = 0; sim < NumberOfSimulations; sim++)
      for(size_t icomp = 0; icomp < SystemComponents[sim].Total_Components; icomp++)
        if(SystemComponents[sim].hasfractionalMolecule[icomp])
          Initialize_WangLandauIteration(SystemComponents[sim].Lambda[icomp]);
  }
  ///////////////////////////////////////////////////////
  // Run the simulations for different boxes IN SERIAL //
  ///////////////////////////////////////////////////////
  for(size_t i = 0; i < Cycles; i++)
  {
    size_t Steps = 1;
    for(size_t sim = 0; sim < NumberOfSimulations; sim++)
    {
      if(Steps < SystemComponents[sim].TotalNumberOfMolecules) 
      {
        Steps = SystemComponents[sim].TotalNumberOfMolecules;
      }
    }
    ////////////////////////////////////////
    // Zhao's note: for debugging purpose //
    ////////////////////////////////////////
    if(SetMaxStep && Steps > MaxStepPerCycle) Steps = MaxStepPerCycle;
    for(size_t j = 0; j < Steps; j++)
    {
      //Draw a random number, if fits, run a Gibbs Box move//
      //Zhao's note: if a Gibbs move is performed, skip the cycle//
      double NVTGibbsRN = Get_Uniform_Random();
      bool GoodForNVTGibbs = false;
      //If no framework atoms for both simulation boxes//
      if(NumberOfSimulations == 2 && SystemComponents[0].Moleculesize[0] == 0 && SystemComponents[1].Moleculesize[0] == 0) GoodForNVTGibbs = true;
      if(GibbsStatistics.DoGibbs && GoodForNVTGibbs)
        if(NVTGibbsRN < GibbsStatistics.GibbsBoxProb) //Zhao's note: for the test, do it at the last step//
        {
          double start = omp_get_wtime();
          printf("Cycle [%zu], Step [%zu], Perform Gibbs Volume Move\n", i, j);
          NVTGibbsMove(SystemComponents, Sims, FF, Energy, GibbsStatistics);
          double end = omp_get_wtime();
          GibbsStatistics.GibbsTime += end - start;
          continue;
        }
      double GibbsXferRN = Get_Uniform_Random();
      if(GibbsStatistics.DoGibbs && GoodForNVTGibbs)
        if(GibbsXferRN < GibbsStatistics.GibbsXferProb)
        {
          //Do a Gibbs Particle Transfer move//
          size_t SelectedComponent = 1;
          printf("Cycle [%zu], Step [%zu], Perform Gibbs Particle Move\n", i, j);
          GibbsParticleTransfer(SystemComponents, Sims, FF, Random, Widom, Energy, SelectedComponent, GibbsStatistics);
          continue;
        }
      for(size_t sim = 0; sim < NumberOfSimulations; sim++)
      {
        RunMoves(i, SystemComponents[sim], Sims[sim], FF, Random, Widom[sim], running_Rosenbluth[sim], SimulationMode);
      }
    }
    for(size_t sim = 0; sim < NumberOfSimulations; sim++)
    {
      //////////////////////////////////////////////
      // SAMPLE (EQUILIBRATION) CBCF BIASING TERM //
      //////////////////////////////////////////////
      if(SimulationMode == EQUILIBRATION && i%50==0)
      {
        for(size_t icomp = 0; icomp < SystemComponents[sim].Total_Components; icomp++)
        { 
          //Try to sample it if there are more CBCF moves performed//
          if(SystemComponents[sim].hasfractionalMolecule[icomp] && !SystemComponents[sim].Tmmc[icomp].DoTMMC)
          {
            Sample_WangLandauIteration(SystemComponents[sim].Lambda[icomp]);
            WLSampled++;
          }
        }
      }

      if(i%500==0)
      {
        for(size_t comp = 0; comp < SystemComponents[sim].Total_Components; comp++)
          if(SystemComponents[sim].Moves[comp].TranslationTotal > 0)
            Update_Max_Translation(SystemComponents[sim], comp);
        for(size_t comp = 0; comp < SystemComponents[sim].Total_Components; comp++)
          if(SystemComponents[sim].Moves[comp].RotationTotal > 0)
            Update_Max_Rotation(SystemComponents[sim], comp);
        for(size_t comp = 0; comp < SystemComponents[sim].Total_Components; comp++)
          if(SystemComponents[sim].Moves[comp].SpecialRotationTotal > 0)
            Update_Max_SpecialRotation(SystemComponents[sim], comp);
      }
      
      if(i % SystemComponents[sim].PrintStatsEvery == 0) Print_Cycle_Statistics(i, SystemComponents[sim], Mode);
      ////////////////////////////////////////////////
      // ADJUST CBCF BIASING FACTOR (EQUILIBRATION) //
      ////////////////////////////////////////////////
      if(i%5000==0 && SimulationMode == EQUILIBRATION)
      {
        for(size_t icomp = 0; icomp < SystemComponents[sim].Total_Components; icomp++)
        if(SystemComponents[sim].hasfractionalMolecule[icomp] && !SystemComponents[sim].Tmmc[icomp].DoTMMC)
        {  
          Adjust_WangLandauIteration(SystemComponents[sim].Lambda[icomp]); 
          WLAdjusted++;
        }
      }
      if(SimulationMode == PRODUCTION)
      {
        //Record values for energy//
        Gather_Averages_Types(SystemComponents[sim].EnergyAverage, Energy[sim].InitialEnergy, Energy[sim].running_energy, i, BlockAverageSize[sim], SystemComponents[sim].Nblock);
        //Record values for Number of atoms//
        for(size_t comp = 0; comp < SystemComponents[sim].Total_Components; comp++)
          Gather_Averages_Types(SystemComponents[sim].Moves[comp].MolAverage, SystemComponents[sim].NumberOfMolecule_for_Component[comp], 0.0, i, BlockAverageSize[sim], SystemComponents[sim].Nblock);
        Gather_Averages_MoveEnergy(SystemComponents[sim], i, BlockAverageSize[sim], SystemComponents[sim].deltaE);
      }
    }
  }
  //print statistics
  if(Cycles > 0)
  {
    for(size_t sim = 0; sim < NumberOfSimulations; sim++)
    {
      if(SimulationMode == EQUILIBRATION) printf("Sampled %zu WangLandau, Adjusted WL %zu times\n", WLSampled, WLAdjusted);
      PrintAllStatistics(SystemComponents[sim], Sims[sim], Cycles, SimulationMode, BlockAverageSize[sim]);
      if(SimulationMode == PRODUCTION)
        Calculate_Overall_Averages_MoveEnergy(SystemComponents[sim], BlockAverageSize[sim]);
    }
    if(GibbsStatistics.DoGibbs)
    {
      PrintGibbs(GibbsStatistics);
    }
  }
}

double Run_Simulation_ForOneBox(int Cycles, Components& SystemComponents, Simulations& Sims, ForceField FF, RandomNumber& Random, WidomStruct Widom, double init_energy, int SimulationMode, bool SetMaxStep, size_t MaxStepPerCycle, Units Constants)
{
  std::vector<size_t>CBCFPerformed(SystemComponents.Total_Components);
  size_t WLSampled = 0; size_t WLAdjusted = 0;

  int BlockAverageSize = 1;
  if(SimulationMode == PRODUCTION)
  {
    BlockAverageSize = Cycles / SystemComponents.Nblock;
    if(Cycles % SystemComponents.Nblock != 0)
      printf("Warning! Number of Cycles cannot be divided by Number of blocks. Residue values go to the last block\n");
  }

  printf("Number of Frameworks: %zu\n", SystemComponents.NumberOfFrameworks);
 
  if(SimulationMode == EQUILIBRATION) //Rezero the TMMC stats at the beginning of the Equilibration cycles//
  {
    //Clear TMMC data in the collection matrix//
    for(size_t comp = 0; comp < SystemComponents.Total_Components; comp++)
      SystemComponents.Tmmc[comp].ClearCMatrix();
  }
  //Clear Rosenbluth weight statistics after Initialization//
  if(SimulationMode == EQUILIBRATION)
  {
    for(size_t comp = 0; comp < SystemComponents.Total_Components; comp++)
      for(size_t i = 0; i < SystemComponents.Nblock; i++)
        SystemComponents.Moves[comp].ClearRosen(i);
  }
  double running_energy = 0.0;
  double running_Rosenbluth = 0.0;
  /////////////////////////////////////////////
  // FINALIZE (PRODUCTION) CBCF BIASING TERM //
  /////////////////////////////////////////////
  //////////////////////////////////////
  // ALSO INITIALIZE AVERAGE ENERGIES //
  //////////////////////////////////////
  if(SimulationMode == PRODUCTION)
  {
    SystemComponents.BookKeepEnergy.resize(SystemComponents.Nblock);
    SystemComponents.BookKeepEnergy_SQ.resize(SystemComponents.Nblock);

    for(size_t icomp = 0; icomp < SystemComponents.Total_Components; icomp++)
      if(SystemComponents.hasfractionalMolecule[icomp] && !SystemComponents.Tmmc[icomp].DoTMMC)
        Finalize_WangLandauIteration(SystemComponents.Lambda[icomp]);
  }

  ///////////////////////////////////////////////////////////////////////
  // FORCE INITIALIZING CBCF BIASING TERM BEFORE INITIALIZATION CYCLES //
  ///////////////////////////////////////////////////////////////////////
  if(SimulationMode == INITIALIZATION && Cycles > 0)
  {
    for(size_t icomp = 0; icomp < SystemComponents.Total_Components; icomp++)
      if(SystemComponents.hasfractionalMolecule[icomp])
        Initialize_WangLandauIteration(SystemComponents.Lambda[icomp]);
  }

  std::string Mode;
  switch(SimulationMode)
  {
    case INITIALIZATION:{Mode = "INITIALIZATION"; break;}
    case EQUILIBRATION: {Mode = "EQUILIBRATION"; break;}
    case PRODUCTION:    {Mode = "PRODUCTION"; break;}
  }

  for(size_t i = 0; i < Cycles; i++)
  {
    size_t Steps = 20;
    if(Steps < SystemComponents.TotalNumberOfMolecules)
    {
      Steps = SystemComponents.TotalNumberOfMolecules;
    }
    //Determine BlockID//
    for(size_t comp = 0; comp < SystemComponents.Total_Components; comp++){
      BlockAverageSize = Cycles / SystemComponents.Nblock;
      if(BlockAverageSize > 0) SystemComponents.Moves[comp].BlockID = i/BlockAverageSize; 
      if(SystemComponents.Moves[comp].BlockID >= SystemComponents.Nblock) SystemComponents.Moves[comp].BlockID--;   }
    ////////////////////////////////////////
    // Zhao's note: for debugging purpose //
    ////////////////////////////////////////
    if(SetMaxStep && Steps > MaxStepPerCycle) Steps = MaxStepPerCycle;
    for(size_t j = 0; j < Steps; j++)
    {
      RunMoves(i, SystemComponents, Sims, FF, Random, Widom, running_Rosenbluth, SimulationMode);
    }
    //////////////////////////////////////////////
    // SAMPLE (EQUILIBRATION) CBCF BIASING TERM //
    //////////////////////////////////////////////
    if(SimulationMode == EQUILIBRATION && i%50==0)
    {
      for(size_t icomp = 0; icomp < SystemComponents.Total_Components; icomp++)
      { //Try to sample it if there are more CBCF moves performed//
        if(SystemComponents.hasfractionalMolecule[icomp] && !SystemComponents.Tmmc[icomp].DoTMMC)
        {
          Sample_WangLandauIteration(SystemComponents.Lambda[icomp]);
          CBCFPerformed[icomp] = SystemComponents.Moves[icomp].CBCFTotal; WLSampled++;
        }
      }
    }

    if(i%500==0)
    {
      for(size_t comp = 0; comp < SystemComponents.Total_Components; comp++)
      {  
        if(SystemComponents.Moves[comp].TranslationTotal > 0)
          Update_Max_Translation(SystemComponents, comp);
        if(SystemComponents.Moves[comp].RotationTotal > 0)
          Update_Max_Rotation(SystemComponents, comp);
        if(SystemComponents.Moves[comp].SpecialRotationTotal > 0)
          Update_Max_SpecialRotation(SystemComponents, comp);
      }
    }
    if(i%SystemComponents.PrintStatsEvery==0) Print_Cycle_Statistics(i, SystemComponents, Mode);
    ////////////////////////////////////////////////
    // ADJUST CBCF BIASING FACTOR (EQUILIBRATION) //
    ////////////////////////////////////////////////
    if(i%5000==0 && SimulationMode == EQUILIBRATION)
    {
      for(size_t icomp = 0; icomp < SystemComponents.Total_Components; icomp++)
        if(SystemComponents.hasfractionalMolecule[icomp] && !SystemComponents.Tmmc[icomp].DoTMMC)//Try not to use CBCFC + TMMC//
        {  Adjust_WangLandauIteration(SystemComponents.Lambda[icomp]); WLAdjusted++;}
    }
    if(SimulationMode == PRODUCTION)
    {
      //Record values for energy//
      Gather_Averages_Types(SystemComponents.EnergyAverage, init_energy, running_energy, i, BlockAverageSize, SystemComponents.Nblock);
      //Record values for Number of atoms//
      for(size_t comp = 0; comp < SystemComponents.Total_Components; comp++)
        Gather_Averages_Types(SystemComponents.Moves[comp].MolAverage, SystemComponents.NumberOfMolecule_for_Component[comp], 0.0, i, BlockAverageSize, SystemComponents.Nblock);
      Gather_Averages_MoveEnergy(SystemComponents, i, BlockAverageSize, SystemComponents.deltaE);
    }
    if(SimulationMode != INITIALIZATION && i > 0)
    {
      for(size_t comp = 0; comp < SystemComponents.Total_Components; comp++)
        if(i % SystemComponents.Tmmc[comp].UpdateTMEvery == 0)
          SystemComponents.Tmmc[comp].AdjustTMBias();
    }
    if(i % SystemComponents.MoviesEvery == 0)//Generate restart file and movies 
      GenerateRestartMovies(SystemComponents, Sims, SystemComponents.PseudoAtoms, 0, SimulationMode);
  }
  //print statistics
  if(Cycles > 0)
  {
    if(SimulationMode == EQUILIBRATION) printf("Sampled %zu WangLandau, Adjusted WL %zu times\n", WLSampled, WLAdjusted);
    PrintAllStatistics(SystemComponents, Sims, Cycles, SimulationMode, BlockAverageSize);
    if(SimulationMode == PRODUCTION)
    {
      Calculate_Overall_Averages_MoveEnergy(SystemComponents, BlockAverageSize);
      Print_Widom_Statistics(SystemComponents, Sims.Box, Constants, 1);
    }
  }
  //At the end of the sim, print a last-step restart and last-step movie
  GenerateRestartMovies(SystemComponents, Sims, SystemComponents.PseudoAtoms, 0, SimulationMode);
  return running_energy;
}
